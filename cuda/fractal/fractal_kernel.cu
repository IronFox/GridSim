#include "hip/hip_runtime.h"
// includes, system
#define _CRT_SECURE_NO_DEPRECATE 1
#define _CRT_NONSTDC_NO_DEPRECATE 1


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  include <windows.h>
#endif

#include <cuda_gl_interop.h>

//#include <cutil.h>

#include "fractal_kernel.h"
#include "../../math/vector.h"
#include "../math/cuda_vector.h"
#include "../math/cuda_composite.h"
#include "../math/cuda_object.h"

/*#include "../../image/image.h"
#include "../../image/converter/png.h"*/

#include "kernel/config.h"


#define DEVICE_RESOURCE(_TYPE_,_NAME_)	const _TYPE_&_NAME_
//#define DEVICE_RESOURCE(_TYPE_,_NAME_)	_TYPE_ _NAME_

#define __DEVICE_CALL__

__device__	void	setHeight(float vector[3],float current_height, float relative_height, const TDeviceSurface&surface, const TContext&context)
{
	#include "kernel/differential_set_height.function"
}


__device__ float	ageAt(unsigned depth)
{
	#include "kernel/age_at.function"
}

__device__ float	getWeight(float h, float average, float variance, float depth)
{
	#include "kernel/get_weight.function"
}


/**
	@brief Queries a random float value in the range [-1,+1] and advances the seed
	@param [inout] seed value
	@return random value
*/
__device__ float	getRandom(int&seed)
{
	#include "kernel/get_random.function"
}


__device__	float	height(const TChannelConfig&channel)
{
	#include "kernel/height.function"
}


__device__ float	getNoise(float distance, const TVertex&p0, const TVertex&p1, const TDeviceSurface&surface, const TContext&context)
{
	#include "kernel/get_noise.function"
}

/* __device__ float	getSmoothStrength(float distance, const TVertex&p0, const TVertex&p1, const TDeviceSurface&surface, const TContext&context)
{
	#include "kernel/get_smooth_strength.function"
}
 */

__device__ void		generate3(const TVertex&p0, const TVertex&p1, const TVertex&p2, TVertex&result, unsigned seed, TDeviceSurface&surface, const TContext&context)
{
	#include "kernel/generate3.function"
}

__device__ void		generate4(const TVertex&p0, const TVertex&p1, const TVertex&p2, const TVertex&p3, TVertex&result, int seed, TDeviceSurface&surface, const TContext&context)
{
	#include "kernel/generate4.function"
}

__device__ void		generate4edge(const TVertex&p0, const TVertex&p1, const TVertex&p2, const TVertex&p3, TVertex&result, int seed, TDeviceSurface&surface, const TContext&context)
{
	#include "kernel/generate4_edge.function"
}

__device__	unsigned	vertexIndex(unsigned x, unsigned y)
{
	return y*(y+1)/2+x;
}


__global__ void generateVertex(TDeviceSurface surface, TDeviceSurface parent, TContext context,const TVertexInfo*info_field, unsigned vertex_count)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	if (linear >= vertex_count)	//rounding error
		return;
	#include "kernel/generate_vertex.function"
}
/*
__global__ void weightVertex(TDeviceSurface surface, TContext context,const TVertexInfo*info_field, unsigned vertex_count)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	if (linear >= vertex_count)	//rounding error
		return;
	#include "kernel/weight_vertex.function"
}
*/

__global__	void	generateNormal(TVertex*vertices, const TVertexInfo*info_field, unsigned vertex_count, unsigned max_row)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	if (linear >= vertex_count)	//rounding error
		return;
		
	#include "kernel/generate_normal.function"


}


__global__	void	detectBoundingBox0(const TVertex*vertices,float*out, unsigned vertex_count)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	if (linear*2 >= vertex_count)	//rounding error
		return;

	const float		*v0 = vertices[linear*2].position,
					*v1 = linear*2+1 >= vertex_count?v0:vertices[linear*2+1].position;
	float	*box = out+linear*6;
	if (v0[0] <  v1[0])
	{
		box[0] = v0[0];
		box[3] = v1[0];
	}
	else
	{
		box[3] = v0[0];
		box[0] = v1[0];
	}
	
	if (v0[1] <  v1[1])
	{
		box[1] = v0[1];
		box[4] = v1[1];
	}
	else
	{
		box[4] = v0[1];
		box[1] = v1[1];
	}

	if (v0[2] <  v1[2])
	{
		box[2] = v0[2];
		box[5] = v1[2];
	}
	else
	{
		box[5] = v0[2];
		box[2] = v1[2];
	}
}

__global__	void	detectBoundingBox1(const float*boxes,float*out, unsigned vertex_count)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	if (linear*2 >= vertex_count)	//rounding error
		return;

	const float	*b0 = boxes+linear*12,
				*b1 = linear*2+1 >= vertex_count?b0:b0+6;
	float	*box = out+linear*6;
	if (b0[0] <  b1[0])
		box[0] = b0[0];
	else
		box[0] = b1[0];
	if (b0[1] <  b1[1])
		box[1] = b0[1];
	else
		box[1] = b1[1];
	if (b0[2] <  b1[2])
		box[2] = b0[2];
	else
		box[2] = b1[2];
		
		
	if (b0[3] >  b1[3])
		box[3] = b0[3];
	else
		box[3] = b1[3];
	if (b0[4] >  b1[4])
		box[4] = b0[4];
	else
		box[4] = b1[4];
	if (b0[5] >  b1[5])
		box[5] = b0[5];
	else
		box[5] = b1[5];
}



__global__	void	detectBoundingRadius0(const TVertex*vertices,float*out, unsigned vertex_count, float3 center)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	if (linear >= vertex_count)	//rounding error
		return;

	const float	*v0 = vertices[linear].position;
	out[linear] = dbSqr(v0[0]-center.x)+dbSqr(v0[1]-center.y)+dbSqr(v0[2]-center.z);
}

__global__	void	detectBoundingRadius1(const float*radi,float*out, unsigned vertex_count)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	if (linear*2 >= vertex_count)	//rounding error
		return;

	const float	r0 = radi[linear*2],
				r1 = linear*2+1 >= vertex_count?r0:radi[linear*2+1];
	out[linear] = fmaxf(r0,r1);

}

__global__	void	mergeEdge(TDeviceSurface this_surface,TDeviceSurface that_surface,TContext context,const TVertexInfo*info_field,const unsigned*edge,const unsigned*neighbor_edge,unsigned edge_length)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	//if (!linear || linear+1 >= edge_length)
	if (linear >= edge_length)
		return;

	#include "kernel/merge_edge.function"

	
	/*this_v.normal[0] = 0;
	this_v.normal[1] = 0;
	this_v.normal[2] = 1;*/
	
	//dvC3(this_v.normal,that_v.normal);
}

__global__	void	copyEdge(TDeviceSurface this_surface,TDeviceSurface that_surface,TContext context,const TVertexInfo*info_field,const unsigned*edge,const unsigned*neighbor_edge,unsigned edge_length, bool copy_to)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	//if (!linear || linear+1 >= edge_length)
	if (linear >= edge_length)
		return;

	#include "kernel/copy_edge.function"

	
	/*this_v.normal[0] = 0;
	this_v.normal[1] = 0;
	this_v.normal[2] = 1;*/
	
	//dvC3(this_v.normal,that_v.normal);
}

__global__	void	generateCover(TCoverVertex*out_vertices,TDeviceSurface surface, TContext context, const unsigned*index_field,unsigned index_count)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	if (linear >= index_count)	//rounding error
		return;
		
	#include "kernel/generate_cover.function"
}

__device__	void	resolveDirection(const TVertex&vertex, const TDeviceSurface&surface, const TContext&context, float v[3])
{
	v[0] = vertex.position[0]+context.sector_size*surface.sector[0];
	v[1] = vertex.position[1]+context.sector_size*surface.sector[1];
	v[2] = vertex.position[2]+context.sector_size*surface.sector[2];
	dvNormalize(v);
}


__global__	void	generateVBO(float*vertex_data,TDeviceSurface surface,TContext context, const TVertexInfo*info_field,const unsigned*index_field, unsigned index_count, unsigned vertex_max)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	if (linear >= index_count)	//rounding error
		return;

	#include "kernel/generate_vbo.function"

}

__device__	float	cloudThickness(const TVertex&v, const TContext&context)
{
	return 1.0;//v.channel.c0*v.channel.c1;
}

__global__	void	generateCloudVBO(float*vertex_data,TDeviceSurface surface,TContext context, const unsigned*index_field, unsigned index_count, unsigned vertex_max)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	if (linear >= index_count/3)	//rounding error
		return;


	float*v = vertex_data+linear*9;
	unsigned	i0 = index_field[linear*3],
				i1 = index_field[linear*3+1],
				i2 = index_field[linear*3+2];
	DEVICE_RESOURCE(TVertex,v0) = surface.vertex[i0];
	DEVICE_RESOURCE(TVertex,v1) = surface.vertex[i1];
	DEVICE_RESOURCE(TVertex,v2) = surface.vertex[i2];
	
	
	float			t0 = cloudThickness(v0,context),
					t1 = cloudThickness(v1,context),
					t2 = cloudThickness(v2,context),
					t = (t0+t1+t2)/3.0f;
	
	dvCenter(v0.position,v1.position,v2.position,v);
	float	h = (v0.height+v1.height+v2.height)/3.0f;
	dvC3(v,v+3);
	setHeight(v,h,1.0f,surface,context);
	setHeight(v+3,h,1.0f+t,surface,context);
	
	float			p0[3],
					p1[3],
					p2[3];
	
	dvC3(v0.position,p0);
	dvC3(v1.position,p1);
	dvC3(v2.position,p2);
	setHeight(p0,v0.height,1.0f+t0,surface,context);
	setHeight(p1,v1.height,1.0f+t1,surface,context);
	setHeight(p2,v2.height,1.0f+t2,surface,context);
	doCalculateTriangleNormal(p0, p1, p2, v+6);
}

__device__	unsigned char*	getTexel3(unsigned char*texel_data, unsigned x, unsigned y, unsigned dimension)
{
	return texel_data+(y*dimension+x)*3;
}

__device__	unsigned char*	getTexel4(unsigned char*texel_data, unsigned x, unsigned y, unsigned dimension)
{
	return texel_data+(y*dimension+x)*4;
}

__device__	unsigned char*	getTexel1(unsigned char*texel_data, unsigned x, unsigned y, unsigned dimension)
{
	return texel_data+(y*dimension+x);
}



__global__ void	generateTexture(unsigned char*texel_data,TDeviceSurface surface,TContext context,const TVertexInfo*info_field,unsigned vertex_count, unsigned dimension)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	if (linear >= vertex_count)	//rounding error
		return;

	#include "kernel/generate_texture.function"

}

__global__ void	generateTextureC(unsigned char*texel_data,TDeviceSurface surface,TContext context,const TVertexInfo*info_field,unsigned vertex_count, unsigned dimension)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	if (linear >= vertex_count)	//rounding error
		return;

	#include "kernel/generate_texture_c.function"

}

__device__ bool _atomicCAS(int*pntr, int compare, int set)
{
	//return atomicCAS(pntr, compare, set)==set;	//wish i could use this...
	
	if (*pntr != compare)
		return false;
	*pntr = set;
	return true;
}



__global__ void cast(TRayIntersection*intersection,TDeviceSurface surface, float3 b_, float3 d_, unsigned*triangle_indices,unsigned triangles)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	if (linear >= triangles)	//rounding error
		return;
		
	float b[3],d[3];
	dvC3(b_,b);
	dvC3(d_,d);
		
		
	#include "kernel/cast.function"


}


__global__ void groundCast(TGroundInfo*ground,TDeviceSurface surface, float3 b_, float3 d_, unsigned*triangle_indices,unsigned triangles)
{
    const unsigned	int	x = blockIdx.x*blockDim.x + threadIdx.x,
						y = blockIdx.y*blockDim.y + threadIdx.y,
						linear = x+y*(blockDim.x*gridDim.x);
	if (linear >= triangles)	//rounding error
		return;

	float b[3],d[3];
	dvC3(b_,b);
	dvC3(d_,d);
	
	#include "kernel/ground_cast.function"

}




namespace CUDA
{
	static const unsigned 	//block_dimension = 14,
							block_x = 128,
							block_y = 1,
							block_size = block_x*block_y;
	
	/**
		Efficiency observations (subjective):
		
		Testing with each 1000 ray lookup operations it turned out that higher thread counts per block are good but the upper limit is unknown.
		One thread per block is definately a bad idea. Device info says 512 threads were fine but resource exceptions are thrown when more than 192 threads are allocated. This correlates to no available specification.
		There may still be some advantage of 192 over 128 threads but the danger that some device may not support that is too great. 128 seems fine for now.
		One dimensional thread blocks seem to be slightly faster than quadratic ones.
	*/

	static	unsigned	ceilHalf(unsigned value)
	{
		unsigned rs = value>>1;
		if (value%2)
			rs++;
		return rs;
	}

	static	unsigned	ceilDiv(unsigned v0, unsigned v1)
	{
		unsigned rs = v0/v1;
		if (v0%v1)
			rs++;
		return rs;
	}


	bool init(float relative_vram_usage)
	{
		if (!Device::initialize(Preference::MostMemory) || !Device::current.totalGlobalMem)
			return false;
		device_channel.memory_limit = (UINT64)((double)Device::current.totalGlobalMem*relative_vram_usage);
		return true;
	}

	void checkStatus(const TCodeLocation&location)
	{
		hipError_t error = hipGetLastError();
		if (error != hipSuccess)
			fatal(location,"Operation failed with error code #"+CString(error)+" ('"+hipGetErrorString(error)+"')");
	}
	

	void generateVertices(THostSurface&surface, THostSurface&parent, const TContext&context,CDeviceArray<TVertexInfo>&device_info_field)
	{
		bool existed = surface.vertex_field.length() == device_info_field.length();
		surface.vertex_field.resize(device_info_field.length(),!Device::emulation);
		if (Device::emulation)
		{
			Emulation::TProcessParameters	parameter;
			parameter.surface = &surface;
			parameter.parent = &parent;
			parameter.context = &context;
			parameter.info_field = device_info_field.hostPointer();
			//parameter.vertex_count = device_info_field.length();
			
			Emulation::process(emuGenerateVertex,parameter,device_info_field.length());
			//Emulation::process(emuWeightVertex,parameter,device_info_field.length());
			
			surface.vertex_field.signalHostMemoryAltered();
		}
		else
		{
			hipGetLastError();
			ARRAY_DEBUG_POINT(k,existed)
			TVertex test = surface.vertex_field[0];
			DEBUG_POINT(k)
			unsigned blocks = ceilDiv(device_info_field.length(),block_size);
			
			dim3 block(block_x, block_y, 1);
			dim3 grid(blocks, 1, 1);

			
			TDeviceSurface	ds;
			((TBaseSurface&)ds) = surface;
			TDeviceSurface	dp;
			((TBaseSurface&)dp) = parent;
			
			ASSERT__(parent.vertex_field.length()==device_info_field.length());
			
			ds.vertex = surface.vertex_field.devicePointer();
			dp.vertex = parent.vertex_field.devicePointer();
			
			
			generateVertex<<< grid, block, 0>>>(ds,dp,context,device_info_field.devicePointer(),device_info_field.length());
			checkStatus(CLOCATION);
			
			/*weightVertex<<< grid, block, 0>>>(ds,context,device_info_field.devicePointer(),device_info_field.length());
			checkStatus(CLOCATION);*/
			surface.vertex_field.signalDeviceMemoryAltered();			
			hipDeviceSynchronize();
			//cout << "gen vertices"<<endl;
			
		}
	}	
	
	

	void generateNormals(THostSurface&surface, const TContext&context,CDeviceArray<TVertexInfo>&device_info_field, unsigned max_row)
	{
		if (Device::emulation)
		{
			Emulation::TProcessParameters	parameter;
			parameter.surface = &surface;
			parameter.context = &context;
			parameter.info_field = device_info_field.hostPointer();
			parameter.max_row = max_row;
			
			Emulation::process(emuGenerateNormal,parameter,device_info_field.length());
			surface.vertex_field.signalHostMemoryAltered();
		}
		else
		{
			hipGetLastError();
			unsigned blocks = ceilDiv(device_info_field.length(),block_size);
			
			dim3 block(block_x, block_y, 1);
			dim3 grid(blocks, 1, 1);

			
			TDeviceSurface	ds;
			((TBaseSurface&)ds) = surface;
			
			//surface.vertex_field.resize(device_info_field.length(),true);
			
			ds.vertex = surface.vertex_field.devicePointer();
			generateNormal<<< grid, block, 0>>>(ds.vertex,device_info_field.devicePointer(),device_info_field.length(),max_row);
			checkStatus(CLOCATION);
			surface.vertex_field.signalDeviceMemoryAltered();
			hipDeviceSynchronize();
			//cout << "gen normals"<<endl;
			
		}
	}	
	


	
	void	getBoundingBox(THostSurface&surface,  float dim[6])
	{
		ASSERT__(!Device::emulation);
		hipGetLastError();
		static CDeviceArray<float>	box_buffer;
		if (box_buffer.size() < surface.vertex_field.length()*12)
			box_buffer.resize(surface.vertex_field.length()*12,true);	//first iteration will create vertex_count/2 boxes, each 6 floats, next iterations will create each half of that. So x12 should suffice
			
			
		unsigned seg_cnt = surface.vertex_field.count();
		unsigned num_runs = ceilHalf(seg_cnt);
		unsigned blocks = ceilDiv(num_runs,block_size);
		
	    dim3 block(block_x, block_y, 1);
	    dim3 grid(blocks, 1, 1);
		
	    detectBoundingBox0<<< grid, block,0 >>>(surface.vertex_field.devicePointer(),box_buffer.devicePointer(),seg_cnt);
		

		float*at = box_buffer.devicePointer();
		while (seg_cnt > 1)
		{
			seg_cnt = ceilHalf(seg_cnt);
			num_runs = ceilHalf(seg_cnt);
			blocks = ceilDiv(num_runs,block_size);
			dim3 block(block_x, block_y, 1);
		    dim3 grid(blocks, 1, 1);
			float*out = at+seg_cnt*6;
			detectBoundingBox1<<< grid, block,0 >>>(at,out,seg_cnt);
			at=out;
		}
		checkStatus(CLOCATION);
		CUDA_ASSERT(hipMemcpy(dim,at,sizeof(float)*6,hipMemcpyDeviceToHost));
		checkStatus(CLOCATION);
	}


	
	float	getBoundingRadius(THostSurface&surface, const float center[3])
	{
		ASSERT__(!Device::emulation);
		hipGetLastError();
		static CDeviceArray<float>	radius_buffer;
		if (radius_buffer.size() < surface.vertex_field.length()*4)
			radius_buffer.resize(surface.vertex_field.length()*4,true);	//first iteration will create vertex_count radi, each 1 float, next iterations will create each half of that. So x4 should suffice
		
		
			
		unsigned seg_cnt = surface.vertex_field.count();
		unsigned num_runs = seg_cnt;
		unsigned blocks = ceilDiv(num_runs,block_size);
		
		
		float3 c = make_float3(center[0],center[1],center[2]);
	    dim3 block(block_x, block_y, 1);
	    dim3 grid(blocks, 1, 1);
		
	    detectBoundingRadius0<<< grid, block,0 >>>(surface.vertex_field.devicePointer(),radius_buffer.devicePointer(),seg_cnt,c);
		


		float*at = radius_buffer.devicePointer();
		while (seg_cnt > 1)
		{
			num_runs = ceilHalf(seg_cnt);
			blocks = ceilDiv(num_runs,block_size);
		
			dim3 block(block_x, block_y, 1);
		    dim3 grid(blocks, 1, 1);
			float*out = at+seg_cnt;
			detectBoundingRadius1<<< grid, block,0 >>>(at,out,seg_cnt);
			at=out;
			
			seg_cnt = ceilHalf(seg_cnt);
		}
		checkStatus(CLOCATION);
		float radius;
		CUDA_ASSERT(hipMemcpy(&radius,at,sizeof(float),hipMemcpyDeviceToHost));
		return sqrt(radius);
	}



	void	mergeEdges(THostSurface&surface, const TContext&context,CDeviceArray<TVertexInfo>&device_info_field, CDeviceArray<unsigned> borders[3], unsigned edge_length)
	{
		//ASSERT__(edge_length>0);
		ASSERT__((surface.flags&(::TBaseSurface::HasData)) != 0);
		if (Device::emulation)
		{
			Emulation::TProcessParameters	parameter;
			parameter.surface = &surface;
			parameter.context = &context;
			parameter.info_field = device_info_field.hostPointer();
			parameter.edge_length = edge_length;
			
			
			for (BYTE k = 0; k < 3; k++)
			{
				ASSERT_EQUAL__(borders[k].length(),edge_length);
				//ASSERT1__(surface.neighbor_link[k].orientation<3,surface.neighbor_link[k].orientation);
				
				if (!surface.neighbor_link[k].surface || !(surface.neighbor_link[k].surface->flags&(::TBaseSurface::HasData)))
				{
					//cout << "warning open edge detected"<<endl;
					continue;
				}
				
				//ASSERT_EQUAL__(&surface, surface.neighbor_link[k].surface->neighbor_link[surface.neighbor_link[k].orientation].surface);
				//ASSERT_EQUAL__(k, surface.neighbor_link[k].surface->neighbor_link[surface.neighbor_link[k].orientation].orientation);
				
				if ((surface.flags&(::TBaseSurface::EdgeMergedOffset<<k)) && (surface.neighbor_link[k].surface->flags&(::TBaseSurface::EdgeMergedOffset<<surface.neighbor_link[k].orientation)))
					continue;	//both adjacent edges have been merged. no need to merge this edge
					
				
				if (!(surface.flags&(::TBaseSurface::EdgeMergedOffset<<k)) && !(surface.neighbor_link[k].surface->flags&(::TBaseSurface::EdgeMergedOffset<<surface.neighbor_link[k].orientation)))
				{
					//DEBUG_POINT(k)
					//cout << "full merge"<<endl;
					//neither this nor the other adjacent edge have been merged. Performing full merge
					parameter.that_surface = surface.neighbor_link[k].surface;
					

					//parameter.copy_to = !(parameter.that_surface.flags & ::TBaseSurface::HasChildren);
					
					parameter.this_border = borders[k].hostPointer();
					parameter.that_border = borders[surface.neighbor_link[k].orientation].hostPointer();
					
					//ASSERT_NOT_NULL__(parameter.surface.vertex);
					//ASSERT_NOT_NULL__(parameter.that_surface.vertex);
				
					/*
						Okay, so far the application mostly (not always) crashed during the following call.
						* all above operations succeeded without problem.
						* surface.neighbor_link[k].orientation is valid.
						* unless something seriously went wrong this_border and that_border are correct
						* with edge_length = 129 Emulation::process will operate linearily, not parallely. Synchronization issues are thus out of the question
						* both segments have data
						* both segments know of each other
						* vertex fields are not NULL
					
					*/
				

					//DEBUG_POINT(k)
					Emulation::process(emuMergeEdge,parameter,edge_length);	//<- crash point
					
				}
				elif (!(surface.flags&(::TBaseSurface::EdgeMergedOffset<<k)))	//this edge has not been merged (and is new compared to the neighboring edge)
				{
					//cout << "left merge"<<endl;
					//DEBUG_POINT(k)
					parameter.that_surface = surface.neighbor_link[k].surface;
					
					parameter.copy_to = false;
					
					parameter.this_border = borders[k].hostPointer();
					parameter.that_border = borders[surface.neighbor_link[k].orientation].hostPointer();
				
					//DEBUG_POINT(k)
					Emulation::process(emuCopyEdge,parameter,edge_length);
				}
				else	// this edge has been merged but not the opposing one
				{
					//cout << "right merge"<<endl;
					//DEBUG_POINT(k)
					parameter.that_surface = surface.neighbor_link[k].surface;
					
					parameter.copy_to = true;
					
					parameter.this_border = borders[k].hostPointer();
					parameter.that_border = borders[surface.neighbor_link[k].orientation].hostPointer();
				
					//DEBUG_POINT(k)
					Emulation::process(emuCopyEdge,parameter,edge_length);
				}

				//DEBUG_POINT(k)
				surface.flags |= (::TBaseSurface::EdgeMergedOffset<<k);
				surface.neighbor_link[k].surface->flags |= (::TBaseSurface::EdgeMergedOffset<<surface.neighbor_link[k].orientation);
				
				
				surface.neighbor_link[k].surface->vertex_field.signalHostMemoryAltered();
				//DEBUG_POINT(k)
			}
			surface.vertex_field.signalHostMemoryAltered();
			//DEBUG_POINT(k)
		}
		else
		{
			hipGetLastError();
			unsigned blocks = ceilDiv(edge_length,block_size);
		
			dim3 block(block_x, block_y, 1);
			dim3 grid(blocks, 1, 1);

			TDeviceSurface	this_surface;
			((TBaseSurface&)this_surface) = surface;
			
			this_surface.vertex = surface.vertex_field.devicePointer();
			
			bool changed = false;
			
			

			for (BYTE k = 0; k < 3; k++)
			{
				ASSERT_EQUAL__(borders[k].length(),edge_length);
				if (!surface.neighbor_link[k].surface || !(surface.neighbor_link[k].surface->flags&(::TBaseSurface::HasData)))
				{
					//cout << "warning open edge detected"<<endl;
					continue;
				}
				if ((surface.flags&(::TBaseSurface::EdgeMergedOffset<<k)) && (surface.neighbor_link[k].surface->flags&(::TBaseSurface::EdgeMergedOffset<<surface.neighbor_link[k].orientation)))
					continue;	//both adjacent edges have been merged. no need to merge this edge
				
				TDeviceSurface	that_surface;
				((TBaseSurface&)that_surface) = *surface.neighbor_link[k].surface;
				ASSERT_EQUAL__(surface.neighbor_link[k].surface->vertex_field.length(),surface.vertex_field.length());
				that_surface.vertex = surface.neighbor_link[k].surface->vertex_field.devicePointer();
				ASSERT_NOT_NULL__(that_surface.vertex);
				
				
				if (!(surface.flags&(::TBaseSurface::EdgeMergedOffset<<k)) && !(surface.neighbor_link[k].surface->flags&(::TBaseSurface::EdgeMergedOffset<<surface.neighbor_link[k].orientation)))
				{
					//bool copy_to = !(that_surface.flags& ::TBaseSurface::HasChildren);
					
					mergeEdge<<< grid, block,0 >>>(this_surface,that_surface,context,device_info_field.devicePointer(),borders[k].devicePointer(),borders[surface.neighbor_link[k].orientation].devicePointer(),edge_length);
					checkStatus(CLOCATION);
					surface.neighbor_link[k].surface->vertex_field.signalDeviceMemoryAltered();
				}
				elif (!(surface.flags&(::TBaseSurface::EdgeMergedOffset<<k)))	//this edge has not been merged (and is new compared to the neighboring edge)
				{
					copyEdge<<< grid, block,0 >>>(this_surface,that_surface,context,device_info_field.devicePointer(),borders[k].devicePointer(),borders[surface.neighbor_link[k].orientation].devicePointer(),edge_length,false);
					checkStatus(CLOCATION);
				}
				else	// this edge has been merged but not the opposing one
				{
					copyEdge<<< grid, block,0 >>>(this_surface,that_surface,context,device_info_field.devicePointer(),borders[k].devicePointer(),borders[surface.neighbor_link[k].orientation].devicePointer(),edge_length,true);
					checkStatus(CLOCATION);
					surface.neighbor_link[k].surface->vertex_field.signalDeviceMemoryAltered();
				}
				surface.flags |= (::TBaseSurface::EdgeMergedOffset<<k);
				surface.neighbor_link[k].surface->flags |= (::TBaseSurface::EdgeMergedOffset<<surface.neighbor_link[k].orientation);
				changed = true;
			}
			if (changed)
			{
				checkStatus(CLOCATION);
				surface.vertex_field.signalDeviceMemoryAltered();
				hipDeviceSynchronize();
			}
			//cout << "merge edges"<<endl;
			
		}
		
		
	}

	void	updateTexture(THostSurface&surface, const TContext&context, CDeviceArray<TVertexInfo>&device_info_field, bool generate_clouds)
	{
		unsigned	range = context.edge_length,
					image_width = range+1+TEXTURE_SEAM_EXTENSION,	//full range plus 1 for diagonal +1 separation row. should suffice but may have to increase
					num_texels = (image_width) * (image_width),
					num_values = num_texels*(generate_clouds?4:3);
		//cout << image_width<<endl;
		//ASSERT1__(!(image_width%2),image_width);	//at the very least it should divisible by 2 //non-power-of-2 from now on
		size_t		data_size = sizeof(GLubyte)*num_values;
	
		if (Device::emulation)
		{
			Emulation::TProcessParameters	parameter;
			parameter.surface = &surface;
			parameter.context = &context;
			parameter.info_field = device_info_field.hostPointer();
			parameter.range = range;
			parameter.image_width = image_width;
			
			static CArray<BYTE>	write_buffer;
			write_buffer.resize(num_values);
			parameter.out = write_buffer;
			
			if (generate_clouds)
				Emulation::process(emuGenerateTextureC,parameter,device_info_field.length());
			else
				Emulation::process(emuGenerateTexture,parameter,device_info_field.length());
			
			
			/*
			CImage	test(image_width,image_width,3);
			test.read(write_buffer.pointer());
			png.saveToFile(test,"test.png");
			exit(0);
			*/
			
			surface.texture.load(write_buffer.pointer(),image_width,image_width,generate_clouds?4:3,false);
		}
		else
		{
			
			static CBufferObject	pbuffer;
				
			checkStatus(CLOCATION);
			
			
			pbuffer.resize(data_size);
			checkStatus(CLOCATION);
			
			
			
			GLubyte*texel_data;
		
			
			pbuffer.registerAndMap(texel_data);
			checkStatus(CLOCATION);
			
			
			TDeviceSurface	this_surface;
			((TBaseSurface&)this_surface) = surface;
			this_surface.vertex = surface.vertex_field.devicePointer();
			
			
			
			unsigned blocks = ceilDiv(device_info_field.length(),block_size);
		
			dim3 block(block_x, block_y, 1);
			dim3 grid(blocks, 1, 1);
			
			if (generate_clouds)
				generateTextureC<<< grid, block,0 >>>(texel_data,this_surface,context,device_info_field.devicePointer(),device_info_field.length(),image_width);
			else
				generateTexture<<< grid, block,0 >>>(texel_data,this_surface,context,device_info_field.devicePointer(),device_info_field.length(),image_width);
			checkStatus(CLOCATION);
			
			pbuffer.release();
			checkStatus(CLOCATION);
			
			/*surface.texture.resize(image_width,image_width);
			checkStatus(CLOCATION);*/

			surface.texture.load(pbuffer,image_width,image_width,generate_clouds?4:3);
			checkStatus(CLOCATION);
			hipDeviceSynchronize();
			
			//cout << "gen texture"<<endl;
		}
	}
	
	
	void	updateVBO(THostSurface&surface, const TContext&context, CDeviceArray<TVertexInfo>&device_info_field, CDeviceArray<unsigned>&device_index_field, unsigned range)
	{
		
		if (Device::emulation)
		{
			Emulation::TProcessParameters	parameter;
			parameter.surface = &surface;
			parameter.context = &context;
			parameter.info_field = device_info_field.hostPointer();
			parameter.index_field = device_index_field.hostPointer();
			parameter.range = range-1;
			
			static CArray<float>	write_buffer;
			write_buffer.resize(device_index_field.length()*9);
			parameter.fout = write_buffer;
			
			Emulation::process(emuGenerateVBO,parameter,device_index_field.length());
			
			surface.vbo.load(write_buffer.pointer(),write_buffer.contentSize());	//4vtx + 3norm + 2tcoord
		}
		else
		{
			//hipGetLastError();
			checkStatus(CLOCATION);
			surface.vbo.resize(device_index_field.length()*9*sizeof(float));	//4vtx + 3norm + 2tcoord
			checkStatus(CLOCATION);
			TDeviceSurface	this_surface;
			((TBaseSurface&)this_surface) = surface;
			this_surface.vertex = surface.vertex_field.devicePointer();
			checkStatus(CLOCATION);
			
		
			unsigned blocks = ceilDiv(device_index_field.length(),block_size);
			dim3 block(block_x, block_y, 1);
			dim3 grid(blocks, 1, 1);

			float*vertex_data;
			
			surface.vbo.registerAndMap(vertex_data);
			
			checkStatus(CLOCATION);
			generateVBO<<< grid, block ,0 >>>(vertex_data,this_surface, context,device_info_field.devicePointer(), device_index_field.devicePointer(),device_index_field.length(), range-1);
			checkStatus(CLOCATION);

			surface.vbo.release();
			checkStatus(CLOCATION);
			hipDeviceSynchronize();
			//cout << "update vbo"<<endl;
			
		}
	}
	
	/*
	void	updateCloudVBO(THostSurface&surface, const TContext&context, CDeviceArray<unsigned>&device_cloud_triangles, unsigned range)
	{
		ASSERT1__(device_cloud_triangles.length()>0 && !(device_cloud_triangles.length()%3),device_cloud_triangles.length());
		TDeviceSurface	this_surface;
		((TBaseSurface&)this_surface) = surface;
		this_surface.vertex = surface.vertex_field.pointer();
		
		surface.cloud_vbo.resize(device_cloud_triangles.length()/3*9*sizeof(float));	//3 base + 3 ceiling + 3 norm
	
		unsigned blocks = ceilDiv(device_cloud_triangles.length()/3,block_size);
		dim3 block(block_dimension, block_dimension, 1);
		dim3 grid(blocks, 1, 1);

		float*vertex_data;
		
		surface.cloud_vbo.registerAndMap(vertex_data);
		
		generateCloudVBO<<< grid, block >>>(vertex_data,this_surface, context, device_cloud_triangles.pointer(),device_cloud_triangles.length(), range-1);
	
		surface.cloud_vbo.release();
	}
	*/
	
	void	retrieveCover(THostSurface&surface, const TContext&context, CDeviceArray<unsigned>&device_index_field, CArray<TCoverVertex>&out_cover)
	{
		if (Device::emulation)
		{
			out_cover.resize(device_index_field.length());
			Emulation::TProcessParameters	parameter;
			parameter.surface = &surface;
			parameter.context = &context;
			parameter.index_field = device_index_field.hostPointer();
			parameter.out_cover = out_cover;
			
			Emulation::process(emuGenerateCover,parameter,device_index_field.length());
		}
		else
		{
			//ASSERT__(!Device::emulation);
			static CDeviceArray<TCoverVertex>	out_vertices;
			
			hipGetLastError();
			out_vertices.resize(device_index_field.length(),true);
			
			TDeviceSurface	this_surface;
			((TBaseSurface&)this_surface) = surface;
			this_surface.vertex = surface.vertex_field.devicePointer();
			
			unsigned blocks = ceilDiv(device_index_field.length(),block_size);
			dim3 block(block_x, block_y, 1);
			dim3 grid(blocks, 1, 1);
			
			generateCover<<< grid, block ,0 >>>(out_vertices.devicePointer(),this_surface, context, device_index_field.devicePointer(),device_index_field.length());
			
			out_vertices.signalDeviceMemoryAltered();
			
			out_vertices.downloadTo(out_cover);
			//cout << "get cover"<<endl;
			
		}
	}
	
	
	bool	rayCast(THostSurface&surface,const float b[3], const float d[3], CDeviceArray<unsigned>&triangles, TRayIntersection&intersection)
	{
		if (Device::emulation)
		{
			Emulation::TProcessParameters	parameter;
			parameter.surface = &surface;
			parameter.b = b;
			parameter.d = d;
			parameter.triangle_indices = triangles.hostPointer();
			parameter.intersection = &intersection;
			Emulation::process(emuCast,parameter,triangles.length()/3);
		}
		else
		{
			static CDeviceArray<TRayIntersection>	device_intersection;
			hipGetLastError();
			device_intersection.resize(1,true);
			hipMemset(device_intersection.devicePointer(), 0, sizeof(TRayIntersection));
			
			TDeviceSurface	this_surface;
			((TBaseSurface&)this_surface) = surface;
			this_surface.vertex = surface.vertex_field.devicePointer();
			
			unsigned blocks = ceilDiv(triangles.length()/3,block_size);
			dim3 block(block_x, block_y, 1);
			dim3 grid(blocks, 1, 1);
			
			cast<<< grid, block ,0 >>>(device_intersection.devicePointer(),this_surface, make_float3(b[0],b[1],b[2]), make_float3(d[0],d[1],d[2]), triangles.devicePointer(),triangles.length()/3);
			device_intersection.signalDeviceMemoryAltered();
			
			intersection = device_intersection[0];
		}
		if (intersection.isset)
		{
			lout << "intersection detected:"<<nl;
			lout << "  fc="<<_toString(intersection.fc)<<nl;
			lout << "  v[0]="<<_toString(intersection.vertex[0].position)<<nl;
			lout << "  v[1]="<<_toString(intersection.vertex[1].position)<<nl;
			lout << "  v[2]="<<_toString(intersection.vertex[2].position)<<nl;
		}
		//cout << (int)intersection.isset<<endl;
		return intersection.isset;
	}
	
	bool	groundQuery(THostSurface&surface, const float b[3], const float down[3], CDeviceArray<unsigned>&triangles, TGroundInfo&ground)
	{
	
		if (Device::emulation)
		{
			Emulation::TProcessParameters	parameter;
			parameter.surface = &surface;
			parameter.b = b;
			parameter.d = down;
			parameter.triangle_indices = triangles.hostPointer();
			parameter.ground = &ground;
			ground.isset = false;
			Emulation::process(emuGroundCast,parameter,triangles.length()/3);
		}
		else
		{
			static CDeviceArray<TGroundInfo>	device_ground;
			hipGetLastError();
			device_ground.resize(1,true);
			hipMemset(device_ground.devicePointer(), 0, sizeof(TGroundInfo));
			
			TDeviceSurface	this_surface;
			((TBaseSurface&)this_surface) = surface;
			this_surface.vertex = surface.vertex_field.devicePointer();
			
			unsigned blocks = ceilDiv(triangles.length()/3,block_size);
			dim3 block(block_x, block_y, 1);
			dim3 grid(blocks, 1, 1);
			
			groundCast<<< grid, block ,0 >>>(device_ground.devicePointer(),this_surface, make_float3(b[0],b[1],b[2]), make_float3(down[0],down[1],down[2]), triangles.devicePointer(),triangles.length()/3);
			
			device_ground.signalDeviceMemoryAltered();
			ground = device_ground[0];
			//cout << "ground query"<<endl;
			
		}
		return ground.isset;	
	}
	
	
	
	
}
